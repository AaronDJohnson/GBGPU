/*  This code was created by Michael Katz.
 *  It is shared under the GNU license (see below).
 *  Creates the structures that hold waveform and interpolation information
 *  for the GPU version of the PhenomHM waveform.
 *
 *
 *  Copyright (C) 2019 Michael Katz
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with with program; see the file COPYING. If not, write to the
 *  Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston,
 *  MA  02111-1307  USA
 */

#include <assert.h>
#include <iostream>
#include <complex>
#include "hip/hip_complex.h"
#include "global.h"

/*
Function for creating ModeContainer on the gpu.
*/

void alloc_waveform(Waveform *wfm)
{
	long i, j, n, k;
	long N;

	N = wfm->N;

  gpuErrchk(hipMalloc(&wfm->params, wfm->NP*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->k, 3*sizeof(double)));


  //gpuErrchk(hipMalloc(&wfm->kdotx, 3*N*sizeof(double))); // vec
  //gpuErrchk(hipMalloc(&wfm->kdotr, 3*3*N*sizeof(double))); // 3x3 mat


  //double *trans_kdotx = new double[3*N];
  //double *trans_kdotr = new double[3*3*N];

  /*for (k=0; k<N; k++){

	for (i=0; i<3; i++)
	{
		//for (j=0; j<3; j++) trans_kdotr[(i*3 + j)*N + k] = 0.;
		//trans_kdotx[i*N + k] = 0.;
	}
}*/

  //gpuErrchk(hipMemcpy(wfm->kdotx, trans_kdotx, 3*N*sizeof(double), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(wfm->kdotr, trans_kdotr, 3*3*N*sizeof(double), hipMemcpyHostToDevice));

  //delete[] trans_kdotr;
  //delete[] trans_kdotx;

  //gpuErrchk(hipMalloc(&wfm->xi, 3*N*sizeof(double)));
  //gpuErrchk(hipMalloc(&wfm->f, 3*N*sizeof(double)));
  //gpuErrchk(hipMalloc(&wfm->fonfs, 3*N*sizeof(double)));

  //double *trans_xi = new double[3*N];
  //double *trans_f = new double[3*N];
  //double *trans_fonfs = new double[3*N];

  /*for (k=0; k<N; k++){
	for (i=0; i<3; i++)
	{
		//trans_xi[i*N + k]    = 0.;
		//trans_f[i*N + k]     = 0.;
		//trans_fonfs[i*N + k] = 0.;
	}
}*/

  //gpuErrchk(hipMemcpy(wfm->xi, trans_xi, 3*N*sizeof(double), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(wfm->f, trans_f, 3*N*sizeof(double), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(wfm->fonfs, trans_fonfs, 3*N*sizeof(double), hipMemcpyHostToDevice));

  //delete[] trans_xi;
  //delete[] trans_f;
  //delete[] trans_fonfs;

	// Polarization basis tensors
  gpuErrchk(hipMalloc(&wfm->eplus, 3*3*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->ecross, 3*3*sizeof(double)));
  //gpuErrchk(hipMalloc(&wfm->dplus, 3*3*N*sizeof(double)));
  //gpuErrchk(hipMalloc(&wfm->dcross, 3*3*N*sizeof(double)));

  /*
  gpuErrchk(hipMalloc(&wfm->r12, 3*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->r21, 3*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->r31, 3*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->r13, 3*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->r23, 3*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->r32, 3*N*sizeof(double)));
  */

  /*
  gpuErrchk(hipMalloc(&wfm->data12, 2*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->data21, 2*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->data31, 2*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->data13, 2*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->data23, 2*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->data32, 2*N*sizeof(double)));
  */

  double *trans_data12 = new double[2*N];
  double *trans_data21 = new double[2*N];
  double *trans_data31 = new double[2*N];
  double *trans_data13 = new double[2*N];
  double *trans_data23 = new double[2*N];
  double *trans_data32 = new double[2*N];

	for (i=0; i<2*N; i++)
	{
		trans_data12[i] = 0.;
		trans_data21[i] = 0.;
		trans_data31[i] = 0.;
		trans_data13[i] = 0.;
		trans_data23[i] = 0.;
		trans_data32[i] = 0.;
	}

  /*
  gpuErrchk(hipMemcpy(wfm->data12, trans_data12, 2*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->data21, trans_data21, 2*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->data31, trans_data31, 2*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->data13, trans_data13, 2*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->data23, trans_data23, 2*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->data32, trans_data32, 2*N*sizeof(double), hipMemcpyHostToDevice));
  */

  delete[] trans_data12;
  delete[] trans_data21;
  delete[] trans_data31;
  delete[] trans_data13;
  delete[] trans_data23;
  delete[] trans_data32;


  gpuErrchk(hipMalloc(&wfm->a12, 2*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->a21, 2*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->a31, 2*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->a13, 2*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->a23, 2*N*sizeof(double)));
  gpuErrchk(hipMalloc(&wfm->a32, 2*N*sizeof(double)));

  double *trans_a12 = new double[2*N];
  double *trans_a21 = new double[2*N];
  double *trans_a31 = new double[2*N];
  double *trans_a13 = new double[2*N];
  double *trans_a23 = new double[2*N];
  double *trans_a32 = new double[2*N];

  for (i=0; i<2*N; i++)
  {
    trans_a12[i] = 0.;
    trans_a21[i] = 0.;
    trans_a31[i] = 0.;
    trans_a13[i] = 0.;
    trans_a23[i] = 0.;
    trans_a32[i] = 0.;
  }

  gpuErrchk(hipMemcpy(wfm->a12, trans_a12, 2*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->a21, trans_a21, 2*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->a31, trans_a31, 2*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->a13, trans_a13, 2*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->a23, trans_a23, 2*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->a32, trans_a32, 2*N*sizeof(double), hipMemcpyHostToDevice));

  delete[] trans_a12;
  delete[] trans_a21;
  delete[] trans_a31;
  delete[] trans_a13;
  delete[] trans_a23;
  delete[] trans_a32;

  //gpuErrchk(hipMalloc(&wfm->TR, 3*3*N*sizeof(double)));
  //gpuErrchk(hipMalloc(&wfm->TI, 3*3*N*sizeof(double)));

  //gpuErrchk(hipMalloc(&wfm->x, 3*N*sizeof(double)));
  //gpuErrchk(hipMalloc(&wfm->y, 3*N*sizeof(double)));
  //gpuErrchk(hipMalloc(&wfm->z, 3*N*sizeof(double)));

  double *trans_eplus = new double[3*3];
  double *trans_ecross = new double[3*3];
  //double *trans_dplus = new double[3*3*N];
  //double *trans_dcross = new double[3*3*N];
  //double *trans_TR = new double[3*3*N];
  //double *trans_TI = new double[3*3*N];
  //double *trans_x = new double[3*N];
  //double *trans_y = new double[3*N];
  //double *trans_z = new double[3*N];
  /*
  double *trans_r12 = new double[3*N];
  double *trans_r21 = new double[3*N];
  double *trans_r31 = new double[3*N];
  double *trans_r13 = new double[3*N];
  double *trans_r23 = new double[3*N];
  double *trans_r32 = new double[3*N];
  */
  for (k=0; k<N; k++){
	for (i=0; i<3; i++)
	{
		for(j=0; j<3; j++)
		{
			trans_eplus[(i*3 + j)]  = 0.;
			trans_ecross[(i*3 + j)] = 0.;
			//trans_dplus[(i*3 + j)*N + k]  = 0.;
			//trans_dcross[(i*3 + j)*N + k] = 0.;
			//trans_TR[(i*3 + j)*N + k]     = 0.;
			//trans_TI[(i*3 + j)*N + k]     = 0.;
		}
		//trans_x[i*N + k]   = 0.;
		//trans_y[i*N + k]   = 0.;
		//trans_z[i*N + k]   = 0.;
    /*
		trans_r12[i*N + k] = 0.;
		trans_r21[i*N + k] = 0.;
		trans_r31[i*N + k] = 0.;
		trans_r13[i*N + k] = 0.;
		trans_r23[i*N + k] = 0.;
		trans_r32[i*N + k] = 0.;
    */
	}
  }

  gpuErrchk(hipMemcpy(wfm->eplus, trans_eplus, 3*3*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->ecross, trans_ecross, 3*3*sizeof(double), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(wfm->dplus, trans_dplus, 3*3*N*sizeof(double), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(wfm->dcross, trans_dcross, 3*3*N*sizeof(double), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(wfm->TR, trans_TR, 3*3*N*sizeof(double), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(wfm->TI, trans_TI, 3*3*N*sizeof(double), hipMemcpyHostToDevice));

  //gpuErrchk(hipMemcpy(wfm->x, trans_x, 3*N*sizeof(double), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(wfm->y, trans_y, 3*N*sizeof(double), hipMemcpyHostToDevice));
  //gpuErrchk(hipMemcpy(wfm->z, trans_z, 3*N*sizeof(double), hipMemcpyHostToDevice));
  /*
  gpuErrchk(hipMemcpy(wfm->r12, trans_r12, 3*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->r21, trans_r21, 3*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->r31, trans_r31, 3*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->r13, trans_r13, 3*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->r23, trans_r23, 3*N*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(wfm->r32, trans_r32, 3*N*sizeof(double), hipMemcpyHostToDevice));
  */

  delete[] trans_eplus;
  delete[] trans_ecross;
  //delete[] trans_dplus;
  //delete[] trans_dcross;
  //delete[] trans_TR;
  //delete[] trans_TI;

  //delete[] trans_x;
  //delete[] trans_y;
  //delete[] trans_z;
  /*
  delete[] trans_r12;
  delete[] trans_r21;
  delete[] trans_r31;
  delete[] trans_r13;
  delete[] trans_r23;
  delete[] trans_r32;
  */
/*
  gpuErrchk(hipMalloc(&wfm->d, 3*3*2*N*sizeof(double)));

  double *trans_d = new double[3*3*2*N];

	for (i=0; i<3; i++)
	{
		for(j=0; j<3; j++)
		{
			for (n=0; n<2*N; n++)
			{
				trans_d[i*3*2*N + j*2*N + n] = 0.;
			}
		}
	}

  gpuErrchk(hipMemcpy(wfm->d, trans_d, 3*3*2*N*sizeof(double), hipMemcpyHostToDevice));

  delete[] trans_d;
*/
	return;
}

void free_waveform(Waveform *wfm){

    gpuErrchk(hipFree(wfm->params));

    gpuErrchk(hipFree(wfm->k));

    gpuErrchk(hipFree(wfm->eplus));
    gpuErrchk(hipFree(wfm->ecross));
    //gpuErrchk(hipFree(wfm->dplus));
    //gpuErrchk(hipFree(wfm->dcross));


    /*
    gpuErrchk(hipFree(wfm->r12));
    gpuErrchk(hipFree(wfm->r21));
    gpuErrchk(hipFree(wfm->r31));
    gpuErrchk(hipFree(wfm->r13));
    gpuErrchk(hipFree(wfm->r23));
    gpuErrchk(hipFree(wfm->r32));
    */

    //gpuErrchk(hipFree(wfm->kdotr));
    //gpuErrchk(hipFree(wfm->kdotx));

    //gpuErrchk(hipFree(wfm->xi));
    //gpuErrchk(hipFree(wfm->f));
    //gpuErrchk(hipFree(wfm->fonfs));

    /*
    gpuErrchk(hipFree(wfm->data12));
    gpuErrchk(hipFree(wfm->data21));
    gpuErrchk(hipFree(wfm->data31));
    gpuErrchk(hipFree(wfm->data13));
    gpuErrchk(hipFree(wfm->data23));
    gpuErrchk(hipFree(wfm->data32));
    */

    gpuErrchk(hipFree(wfm->a12));
    gpuErrchk(hipFree(wfm->a21));
    gpuErrchk(hipFree(wfm->a31));
    gpuErrchk(hipFree(wfm->a13));
    gpuErrchk(hipFree(wfm->a23));
    gpuErrchk(hipFree(wfm->a32));

    //gpuErrchk(hipFree(wfm->x));
    //gpuErrchk(hipFree(wfm->y));
    //gpuErrchk(hipFree(wfm->z));

    //gpuErrchk(hipFree(wfm->TR));
    //gpuErrchk(hipFree(wfm->TI));

    //gpuErrchk(hipFree(wfm->d));

}
