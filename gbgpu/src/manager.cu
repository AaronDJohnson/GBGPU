#include "hip/hip_runtime.h"
/*  This code was created by Michael Katz.
 *  It is shared under the GNU license (see below).
 *  This is the central piece of code. This file implements a class
 *  that takes data in on the cpu side, copies
 *  it to the gpu, and exposes functions that let
 *  you perform actions with the GPU.
 *
 *  This class will get translated into python via cython.
 *
 *
 *
 *  Copyright (C) 2019 Michael Katz
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with with program; see the file COPYING. If not, write to the
 *  Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston,
 *  MA  02111-1307  USA
 */

#ifdef __HIPCC__

#include "hip/hip_complex.h"
#include "hipblas.h"
#include <kernel.cu>
#include "createGPUHolders.cu"
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
//#include <hip/hip_runtime.h>

#else

#include <kernel.cpp>

#endif

#include <manager.hh>
#include <assert.h>
#include <iostream>
#include <complex>

#include "omp.h"
//#include "cuda_complex.hpp"
// TODO: CUTOFF PHASE WHEN IT STARTS TO GO BACK UP!!!

using namespace std;

#define BATCH 1

#ifdef __HIPCC__
void print_mem_info(){
        // show memory usage of GPU

        hipError_t cuda_status;

        size_t free_byte ;

        size_t total_byte ;

        cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;

        if ( hipSuccess != cuda_status ){

            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );

            exit(1);

        }



        double free_db = (double)free_byte ;

        double total_db = (double)total_byte ;

        double used_db = total_db - free_db ;

        printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

            used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}

#endif

GBGPU::GBGPU (
    int data_stream_length_,
    double *data_freqs_,
    long ptr_data_channel1_,
    long ptr_data_channel2_,
    long ptr_data_channel3_, int N_,
    int nwalkers_,
    int ndevices_,
    double Tobs_,
    double dt_,
    int NP_){

      Tobs = Tobs_;
      dt = dt_;
      NP = NP_;


    #pragma omp parallel
    {
      if (omp_get_thread_num() == 1) printf("NUM OMP THREADS: %d\n", omp_get_num_threads());
    }


    data_freqs = data_freqs_;
    N = N_;
    data_stream_length = data_stream_length_;
    /*channel1_ASDinv = channel1_ASDinv_;
    channel2_ASDinv = channel2_ASDinv_;
    channel3_ASDinv = channel3_ASDinv_;
    data_channel1 = data_channel1_;
    data_channel2 = data_channel2_;
    data_channel3 = data_channel3_;*/
    nwalkers = nwalkers_;
    ndevices = ndevices_;

    ndevices = ndevices_;

      gpuErrchk(hipMalloc(&d_data_freqs, data_stream_length*sizeof(double)));

      gpuErrchk(hipMemcpy(d_data_freqs, data_freqs, data_stream_length*sizeof(double), hipMemcpyHostToDevice));

      /*gpuErrchk(hipMalloc(&d_data_channel1, data_stream_length*sizeof(agcmplx)));

      gpuErrchk(hipMalloc(&d_data_channel2, data_stream_length*sizeof(agcmplx)));

      gpuErrchk(hipMalloc(&d_data_channel3, data_stream_length*sizeof(agcmplx)));

      gpuErrchk(hipMalloc(&d_channel1_ASDinv, data_stream_length*sizeof(double)));

      gpuErrchk(hipMalloc(&d_channel2_ASDinv, data_stream_length*sizeof(double)));

      gpuErrchk(hipMalloc(&d_channel3_ASDinv, data_stream_length*sizeof(double)));*/

      h_wfm = new Waveform[nwalkers];

      for (int i=0; i<nwalkers; i++){
          h_wfm[i].NP = NP;
          h_wfm[i].T = Tobs;
          h_wfm[i].N = N;
          N = h_wfm[i].N;
          alloc_waveform(&h_wfm[i]);
      }
      gpuErrchk(hipMalloc(&wfm, nwalkers*sizeof(Waveform)));



      gpuErrchk(hipMemcpy(wfm, h_wfm, nwalkers*sizeof(Waveform), hipMemcpyHostToDevice));

      //gpuErrchk(hipMalloc(&X_buffer, 2*N*nwalkers*sizeof(double)));
      //gpuErrchk(hipMalloc(&Y_buffer, 2*N*nwalkers*sizeof(double)));
      //gpuErrchk(hipMalloc(&Z_buffer, 2*N*nwalkers*sizeof(double)));

      gpuErrchk(hipMalloc(&data12, 2*N*nwalkers*sizeof(double)));
      gpuErrchk(hipMalloc(&data21, 2*N*nwalkers*sizeof(double)));
      gpuErrchk(hipMalloc(&data13, 2*N*nwalkers*sizeof(double)));
      gpuErrchk(hipMalloc(&data31, 2*N*nwalkers*sizeof(double)));
      gpuErrchk(hipMalloc(&data23, 2*N*nwalkers*sizeof(double)));
      gpuErrchk(hipMalloc(&data32, 2*N*nwalkers*sizeof(double)));

      //gpuErrchk(hipMalloc(&XLS, 2*data_stream_length*sizeof(double)));
      //gpuErrchk(hipMalloc(&YLS, 2*data_stream_length*sizeof(double)));
      //gpuErrchk(hipMalloc(&ZLS, 2*data_stream_length*sizeof(double)));

      XLS = (double *)ptr_data_channel1_;
      YLS = (double *)ptr_data_channel2_;
      ZLS = (double *)ptr_data_channel3_;

      //gpuErrchk(hipMalloc(&XSL, 2*data_stream_length*sizeof(double)));
      //gpuErrchk(hipMalloc(&YSL, 2*data_stream_length*sizeof(double)));
      //gpuErrchk(hipMalloc(&ZSL, 2*data_stream_length*sizeof(double)));

      gpuErrchk(hipMalloc(&d_params, NP*nwalkers*sizeof(double)));


      // for likelihood
      // --------------
      stat = hipblasCreate(&handle);
      if (stat != HIPBLAS_STATUS_SUCCESS) {
              printf ("CUBLAS initialization failed\n");
              exit(0);
          }

  if (hipfftPlan1d(&plan, h_wfm->N, HIPFFT_Z2Z, nwalkers) != HIPFFT_SUCCESS){
        	fprintf(stderr, "CUFFT error: Plan creation failed");
        	return;	}

  //GBGPU::input_data(data_freqs, data_channel1,
    //                    data_channel2, data_channel3,
      //                  channel1_ASDinv, channel2_ASDinv, channel3_ASDinv, data_stream_length);

}

/*
void GBGPU::input_data(double *data_freqs_, cmplx *data_channel1_,
                          cmplx *data_channel2_, cmplx *data_channel3_,
                          double *channel1_ASDinv_, double *channel2_ASDinv_,
                          double *channel3_ASDinv_, int data_stream_length_){

    assert(data_stream_length_ == data_stream_length);

        gpuErrchk(hipMemcpy(d_data_freqs, data_freqs_, data_stream_length*sizeof(double), hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_data_channel1, data_channel1_, data_stream_length*sizeof(agcmplx), hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_data_channel2, data_channel2_, data_stream_length*sizeof(agcmplx), hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_data_channel3, data_channel3_, data_stream_length*sizeof(agcmplx), hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_channel1_ASDinv, channel1_ASDinv_, data_stream_length*sizeof(double), hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_channel2_ASDinv, channel2_ASDinv_, data_stream_length*sizeof(double), hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_channel3_ASDinv, channel3_ASDinv_, data_stream_length*sizeof(double), hipMemcpyHostToDevice));
}
*/

__global__
void fill_params(Waveform *wfm_trans, double *params, int nwalkers, int NP)
{
    Waveform *wfm;
  for (int walker_i = blockIdx.x * blockDim.x + threadIdx.x;
       walker_i < nwalkers;
       walker_i += blockDim.x * gridDim.x){
         wfm = &wfm_trans[walker_i];

         for (int i=0; i<NP; i++) wfm->params[0] = params[walker_i*NP + i];
         wfm->q  = (long)(wfm->params[0]);

  }
}


void GBGPU::Fast_GB(double *params_){//,double *XLS, double *YLS, double *ZLS,double* XSL, double* YSL, double* ZSL){

    gpuErrchk(hipMemcpy(d_params, params_, nwalkers*NP*sizeof(double), hipMemcpyHostToDevice));

      //gpuErrchk(hipMemcpy(wfm, h_wfm, nwalkers*sizeof(Waveform), hipMemcpyHostToDevice));

      int NUM_THREADS = 256;
      int num_blocks_basis_tensors = std::ceil((nwalkers + NUM_THREADS -1)/NUM_THREADS);

      fill_params<<<num_blocks_basis_tensors,NUM_THREADS>>>(wfm, d_params, nwalkers,NP);
      hipDeviceSynchronize();
      gpuErrchk(hipGetLastError());

    	get_basis_tensors<<<num_blocks_basis_tensors,NUM_THREADS>>>(wfm, nwalkers);      //  Tensor construction for building slowly evolving LISA response
      hipDeviceSynchronize();
      gpuErrchk(hipGetLastError());

      int num_blocks = std::ceil((h_wfm->N + NUM_THREADS -1)/NUM_THREADS);
      dim3 gridDim(num_blocks, nwalkers);
      GenWave<<<gridDim, NUM_THREADS>>>(wfm, h_wfm->N, nwalkers,
                                         data12, data21, data13, data31, data23, data32);
      hipDeviceSynchronize();
      gpuErrchk(hipGetLastError());

      fft_data(data12, data21, data13, data31, data23, data32, plan, nwalkers);

      unpack_data_1<<<gridDim, NUM_THREADS>>>(wfm, data12, data21, data13, data31, data23, data32, nwalkers);
      hipDeviceSynchronize();
      gpuErrchk(hipGetLastError());

      /*unpack_data_2<<<gridDim, NUM_THREADS>>>(wfm, nwalkers);
      hipDeviceSynchronize();
      gpuErrchk(hipGetLastError());*/

      XYZ_wrap<<<gridDim, NUM_THREADS>>>(wfm, nwalkers, N, dt, Tobs, XLS, YLS, ZLS);
      hipDeviceSynchronize();
      gpuErrchk(hipGetLastError());
}


void GBGPU::Likelihood(double *likelihood){

  double d_h = 0.0;
  double h_h = 0.0;
  char * status;
  double res;
  hipDoubleComplex result;
  hipblasStatus_t stat;

          // get data - template terms
           stat = hipblasZdotc(handle, data_stream_length,
                   (hipDoubleComplex*)XLS, 1,
                   (hipDoubleComplex*)d_data_channel1, 1,
                   &result);

            d_h += hipCreal(result);

            stat = hipblasZdotc(handle, data_stream_length,
                    (hipDoubleComplex*)YLS, 1,
                    (hipDoubleComplex*)d_data_channel2, 1,
                    &result);
             d_h += hipCreal(result);

             stat = hipblasZdotc(handle, data_stream_length,
                     (hipDoubleComplex*)ZLS, 1,
                     (hipDoubleComplex*)d_data_channel3, 1,
                     &result);

           d_h += hipCreal(result);

           //<h|h>
           stat = hipblasZdotc(handle, data_stream_length,
                   (hipDoubleComplex*)XLS, 1,
                   (hipDoubleComplex*)XLS, 1,
                   &result);

            h_h += hipCreal(result);

            stat = hipblasZdotc(handle, data_stream_length,
                    (hipDoubleComplex*)YLS, 1,
                    (hipDoubleComplex*)YLS, 1,
                    &result);

             h_h += hipCreal(result);

             stat = hipblasZdotc(handle, data_stream_length,
                     (hipDoubleComplex*)ZLS, 1,
                     (hipDoubleComplex*)ZLS, 1,
                     &result);

           h_h += hipCreal(result);

           // reverse TODO: do I need it?
            stat = hipblasZdotc(handle, data_stream_length,
                    (hipDoubleComplex*)XSL, 1,
                    (hipDoubleComplex*)d_data_channel1, 1,
                    &result);

             d_h += hipCreal(result);

             stat = hipblasZdotc(handle, data_stream_length,
                     (hipDoubleComplex*)YSL, 1,
                     (hipDoubleComplex*)d_data_channel2, 1,
                     &result);
              d_h += hipCreal(result);

              stat = hipblasZdotc(handle, data_stream_length,
                      (hipDoubleComplex*)ZSL, 1,
                      (hipDoubleComplex*)d_data_channel3, 1,
                      &result);

            d_h += hipCreal(result);

            //<h|h>
            stat = hipblasZdotc(handle, data_stream_length,
                    (hipDoubleComplex*)XSL, 1,
                    (hipDoubleComplex*)XSL, 1,
                    &result);

             h_h += hipCreal(result);

             stat = hipblasZdotc(handle, data_stream_length,
                     (hipDoubleComplex*)YSL, 1,
                     (hipDoubleComplex*)YSL, 1,
                     &result);

              h_h += hipCreal(result);

              stat = hipblasZdotc(handle, data_stream_length,
                      (hipDoubleComplex*)ZSL, 1,
                      (hipDoubleComplex*)ZSL, 1,
                      &result);

            h_h += hipCreal(result);
               //printf("channel3 h_h: %e\n", hipCreal(result));
           likelihood[0] = 4*d_h;
           likelihood[1] = 4*h_h;
}

/*
Destructor
*/
GBGPU::~GBGPU() {

  gpuErrchk(hipFree(d_data_freqs));
/*  gpuErrchk(hipFree(d_data_channel1));
  gpuErrchk(hipFree(d_data_channel2));
  gpuErrchk(hipFree(d_data_channel3));

  gpuErrchk(hipFree(d_channel1_ASDinv));
  gpuErrchk(hipFree(d_channel2_ASDinv));
  gpuErrchk(hipFree(d_channel3_ASDinv));*/
  free_waveform(wfm);
  //free(h_wfm);
  delete[] h_wfm;
  hipblasDestroy(handle);

//gpuErrchk(hipFree(X_buffer));
//gpuErrchk(hipFree(Y_buffer));
//gpuErrchk(hipFree(Z_buffer));

gpuErrchk(hipFree(d_params));

//gpuErrchk(hipFree(XLS));
//gpuErrchk(hipFree(YLS));
//gpuErrchk(hipFree(ZLS));

gpuErrchk(hipFree(XSL));
gpuErrchk(hipFree(YSL));
gpuErrchk(hipFree(ZSL));

gpuErrchk(hipFree(data12));
gpuErrchk(hipFree(data21));
gpuErrchk(hipFree(data13));
gpuErrchk(hipFree(data31));
gpuErrchk(hipFree(data23));
gpuErrchk(hipFree(data32));

hipfftDestroy(plan);
gpuErrchk(hipFree(wfm));
}

int GetDeviceCount(){
    int num_device_check;
    #ifdef __HIPCC__
    hipError_t cuda_status = hipGetDeviceCount(&num_device_check);
    if (hipSuccess != cuda_status) num_device_check = 0;
    #else
    num_device_check = 0;
    #endif
    printf("NUMBER OF DEVICES: %d\n", num_device_check);
    return num_device_check;
}
