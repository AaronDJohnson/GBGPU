#include "hip/hip_runtime.h"
/*  This code was edited by Michael Katz. It is originally from the LAL library.
 *  The original copyright and license is shown below. Michael Katz has edited
 *  the code for his purposes and removed dependencies on the LAL libraries. The code has been confirmed to match the LAL version.
 *  This code is distrbuted under the same GNU license it originally came with.
 *  The comments in the code have been left generally the same. A few comments
 *  have been made for the newer functions added.

 * This code is adjusted for usage in CUDA. Refer to PhenomHM.cpp for comments.


 *  Copyright (C) 2017 Sebastian Khan, Francesco Pannarale, Lionel London
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with with program; see the file COPYING. If not, write to the
 *  Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston,
 *  MA  02111-1307  USA
 */

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include "cuda_complex.hpp"
#include "Constants.h"
#include "LISA.h"
#include <assert.h>

#include "global.h"

#ifdef __HIPCC__
#else
#include "omp.h"
#endif

#include <hipfft/hipfft.h>

__device__
void set_const_trans(Waveform *wfm)
{
	double amp, cosiota;
	double Aplus, Across;
	double psi;
	double sinps, cosps;

	amp      = exp(wfm->params[3]);
	cosiota  = wfm->params[4];
	psi      = wfm->params[5];

	//Calculate GW polarization amplitudes
	Aplus  = amp*(1. + cosiota*cosiota);
	// Aplus  = -amp*(1. + cosiota*cosiota);
	Across = -2.0*amp*cosiota;
	//Across = 2.0*amp*cosiota;

	//Calculate cos and sin of polarization
	cosps = cos(2.*psi);
	sinps = sin(2.*psi);

	//Calculate constant pieces of transfer functions
	wfm->DPr    =  Aplus*cosps;
	wfm->DPi    = -Across*sinps;
	wfm->DCr    = -Aplus*sinps;
	wfm->DCi    = -Across*cosps;

	return;
}

__device__
void get_basis_vecs(double *params, double *u, double *v, double *k)
{
	long i;

	double phi;
	double costh, sinth, cosph, sinph;

	for (i=0; i<3; i++)
	{
		u[i] = 0.;
		v[i] = 0.;
		k[i] = 0.;
	}

	phi	  = params[2];
	costh = params[1];

	sinth = sqrt(1.0-costh*costh);

	cosph = cos(phi);
	sinph = sin(phi);

	u[0] =  costh*cosph;  u[1] =  costh*sinph;  u[2] = -sinth;
	v[0] =  sinph;        v[1] = -cosph;        v[2] =  0.;
	k[0] = -sinth*cosph;  k[1] = -sinth*sinph;  k[2] = -costh;

	return;
}

__global__
void get_basis_tensors(Waveform *wfm_trans, int nwalkers)
{

	Waveform * wfm;
	long i, j;

	 // GW basis vectors

	double u[3];
	double v[3];

	for (int walker_i = blockIdx.x * blockDim.x + threadIdx.x;
			 walker_i < nwalkers;
			 walker_i += blockDim.x * gridDim.x){

	wfm = &wfm_trans[walker_i];

	set_const_trans(wfm);  // set the constant pieces of transfer function

	get_basis_vecs(wfm->params, u, v, wfm->k); //Gravitational Wave source basis vectors

	//GW polarization basis tensors
	for(i=0;i<3;i++)
	{
		for(j=0;j<3;j++)
		{
			//wfm->eplus[i][j]  = u[i]*u[j] - v[i]*v[j];
			wfm->eplus[i*3 + j]  = v[i]*v[j] - u[i]*u[j];
			wfm->ecross[i*3 + j] = u[i]*v[j] + v[i]*u[j];
			//wfm->ecross[i][j] = -u[i]*v[j] - v[i]*u[j];
		}
	}
}

	return;
}

__device__
void spacecraft(double t, double *x, double *y, double *z, int n, int N)
{
	double alpha;
	double beta1, beta2, beta3;
	double sa, sb, ca, cb;

	alpha = 2.*M_PI*fm*t + kappa;

	beta1 = 0. + lambda;
	beta2 = 2.*M_PI/3. + lambda;
	beta3 = 4.*M_PI/3. + lambda;

	sa = sin(alpha);
	ca = cos(alpha);

	sb = sin(beta1);
	cb = cos(beta1);
	x[0] = AU*ca + AU*ec*(sa*ca*sb - (1. + sa*sa)*cb);
	y[0] = AU*sa + AU*ec*(sa*ca*cb - (1. + ca*ca)*sb);
	z[0] = -SQ3*AU*ec*(ca*cb + sa*sb);

	sb = sin(beta2);
	cb = cos(beta2);
	x[1] = AU*ca + AU*ec*(sa*ca*sb - (1. + sa*sa)*cb);
	y[1] = AU*sa + AU*ec*(sa*ca*cb - (1. + ca*ca)*sb);
	z[1] = -SQ3*AU*ec*(ca*cb + sa*sb);

	sb = sin(beta3);
	cb = cos(beta3);
	x[2] = AU*ca + AU*ec*(sa*ca*sb - (1. + sa*sa)*cb);
	y[2] = AU*sa + AU*ec*(sa*ca*cb - (1. + ca*ca)*sb);
	z[2] = -SQ3*AU*ec*(ca*cb + sa*sb);

	return;
}

__device__
void calc_xi_f(Waveform *wfm, double t, int n, int N, double *x, double *y, double *z, double *xi, double *fonfs)
{
	long i;

	double f0, dfdt_0, d2fdt2_0;

	double kdotx[3];
	double f[3];

	f0       = wfm->params[0]/wfm->T;
	if (wfm->NP > 7) dfdt_0   = wfm->params[7]/wfm->T/wfm->T;
	if (wfm->NP > 8) d2fdt2_0 = wfm->params[8]/wfm->T/wfm->T/wfm->T;

	spacecraft(t, x, y, z, n, N); // Calculate position of each spacecraft at time t

	for(i=0; i<3; i++)
	{
		kdotx[i] = (x[i]*wfm->k[0] + y[i]*wfm->k[1] + z[i]*wfm->k[2])/C;
		//Wave arrival time at spacecraft i
		xi[i]    = t - kdotx[i];
		//FIXME
		//xi[i]    = t + kdotx[i];
		//First order approximation to frequency at spacecraft i
		f[i]     = f0;
		if (wfm->NP > 7) f[i] += dfdt_0*xi[i];
		if (wfm->NP > 8) f[i] += 0.5*d2fdt2_0*xi[i]*xi[i];

		//Ratio of true frequency to transfer frequency
		fonfs[i] = f[i]/fstar;
	}

	return;
}

__device__
void calc_sep_vecs(Waveform *wfm, int n, int N, double *x, double *y, double *z, double *r12, double *r21, double *r13, double *r31, double *r23, double *r32)
{
	long i;

	//Unit separation vector from spacecrafts i to j
	r12[0] = (x[1] - x[0])/Larm;
	r13[0] = (x[2] - x[0])/Larm;
	r23[0] = (x[2] - x[1])/Larm;
	r12[1] = (y[1] - y[0])/Larm;
	r13[1] = (y[2] - y[0])/Larm;
	r23[1] = (y[2] - y[1])/Larm;
	r12[2] = (z[1] - z[0])/Larm;
	r13[2] = (z[2] - z[0])/Larm;
	r23[2] = (z[2] - z[1])/Larm;

	//Make use of symmetry
	for(i=0; i<3; i++)
	{
		r21[i] = -r12[i];
		r31[i] = -r13[i];
		r32[i] = -r23[i];
	}
	return;
}

__device__
void calc_d_matrices(Waveform *wfm, int n, int N, double *dcross, double *dplus, double *r12, double *r21, double *r13, double *r31, double *r23, double *r32)
{
	long i, j;

	//Zero arrays to be summed
	dplus [(0*3 + 1)] = 0.0;
	dplus [(0*3 + 2)] = 0.0;
	dplus [(1*3 + 0)] = 0.;
	dplus [(1*3 + 2)] = 0.0;
	dplus [(2*3 + 0)] = 0.0;
	dplus [(2*3 + 1)] = 0.;
	dcross[(0*3 + 1)] = 0.0;
	dcross[(0*3 + 2)] = 0.0;
	dcross[(1*3 + 0)] = 0.;
	dcross[(1*3 + 2)] = 0.0;
	dcross[(2*3 + 0)] = 0.0;
	dcross[(2*3 + 1)] = 0.;

	//Convenient quantities d+ & dx
	for(i=0; i<3; i++)
	{
		for(j=0; j<3; j++)
		{
			dplus [(0*3 + 1)] += r12[i]*r12[j]*wfm->eplus[i*3 + j];
			dcross[(0*3 + 1)] += r12[i]*r12[j]*wfm->ecross[i*3 + j];
			dplus [(1*3 + 2)] += r23[i]*r23[j]*wfm->eplus[i*3 + j];
			dcross[(1*3 + 2)] += r23[i]*r23[j]*wfm->ecross[i*3 + j];
			dplus [(0*3 + 2)] += r13[i]*r13[j]*wfm->eplus[i*3 + j];
			dcross[(0*3 + 2)] += r13[i]*r13[j]*wfm->ecross[i*3 + j];
		}
	}
	//Makng use of symmetry
	dplus[(1*3 + 0)] = dplus[(0*3 + 1)];  dcross[(1*3 + 0)] = dcross[(0*3 + 1)];
	dplus[(2*3 + 1)] = dplus[(1*3 + 2)];  dcross[(2*3 + 1)] = dcross[(1*3 + 2)];
	dplus[(2*3 + 0)] = dplus[(0*3 + 2)];  dcross[(2*3 + 0)] = dcross[(0*3 + 2)];

	return;
}


__device__
void calc_kdotr(Waveform *wfm, int n, int N, double *kdotr, double *r12, double *r21, double *r13, double *r31, double *r23, double *r32)
{
	long i;

	//Zero arrays to be summed
	kdotr[(0*3 + 1)] = 0.0;
	kdotr[(0*3 + 2)] = 0.0;
	kdotr[(1*3 + 0)] = 0.;
	kdotr[(1*3 + 2)] = 0.0;
	kdotr[(2*3 + 0)] = 0.0;
	kdotr[(2*3 + 1)] = 0.;

	for(i=0; i<3; i++)
	{
		kdotr[(0*3 + 1)] += wfm->k[i]*r12[i];
		kdotr[(0*3 + 2)] += wfm->k[i]*r13[i];
		kdotr[(1*3 + 2)] += wfm->k[i]*r23[i];
	}

	//Making use of antisymmetry
	kdotr[(1*3 + 0)] = -kdotr[(0*3 + 1)];
	kdotr[(2*3 + 0)] = -kdotr[(0*3 + 2)];
	kdotr[(2*3 + 1)] = -kdotr[(1*3 + 2)];

	return;
}


__device__

void get_transfer(Waveform *wfm, double t, int n, int N, double *kdotr, double *TR, double *TI, double *dplus, double *dcross,
									double *xi, double *fonfs)
{
	long i, j;
	long q;

	double tran1r, tran1i;
	double tran2r, tran2i;
	double aevol;			// amplitude evolution factor
	double arg1, arg2, sinc;
	double f0, dfdt_0, d2fdt2_0;
	double df, phi0;

	f0       = wfm->params[0]/wfm->T;
	phi0     = wfm->params[6];

	if (wfm->NP > 7) dfdt_0   = wfm->params[7]/wfm->T/wfm->T;
 	if (wfm->NP > 8) d2fdt2_0 = wfm->params[8]/wfm->T/wfm->T/wfm->T;

	q  = wfm->q;
	df = PI2*(((double)q)/wfm->T);

	for(i=0; i<3; i++)
	{
		for(j=0; j<3; j++)
		{
			if(i!=j)
			{
				//Argument of transfer function
				// FIXME
				//arg1 = 0.5*fonfs[i]*(1. - kdotr[i][j]);
				arg1 = 0.5*fonfs[i]*(1. + kdotr[(i*3 + j)]);

				//Argument of complex exponentials
				arg2 = PI2*f0*xi[i] + phi0 - df*t;

				if (wfm->NP > 7) arg2 += M_PI*dfdt_0*xi[i]*xi[i];
				if (wfm->NP > 8) arg2 += M_PI*d2fdt2_0*xi[i]*xi[i]*xi[i]/3.0 ;

				//Transfer function
				sinc = 0.25*sin(arg1)/arg1;

				//Evolution of amplitude
				aevol = 1.0;
				if (wfm->NP > 7) aevol += 0.66666666666666666666*dfdt_0/f0*xi[i];

				///Real and imaginary pieces of time series (no complex exponential)
				tran1r = aevol*(dplus[(i*3 + j)]*wfm->DPr + dcross[(i*3 + j)]*wfm->DCr);
				tran1i = aevol*(dplus[(i*3 + j)]*wfm->DPi + dcross[(i*3 + j)]*wfm->DCi);

				//Real and imaginry components of complex exponential
				tran2r = cos(arg1 + arg2);
				tran2i = sin(arg1 + arg2);

				//Real & Imaginary part of the slowly evolving signal
				TR[(i*3 + j)] = sinc*(tran1r*tran2r - tran1i*tran2i);
				TI[(i*3 + j)] = sinc*(tran1r*tran2i + tran1i*tran2r);
			}
		}
	}

	return;
}


__device__

void fill_time_series(int walker_i, int n, int N, double *TR, double *TI,
											double *data12, double *data21, double *data13,
										  double *data31, double *data23, double *data32)
{
	data12[(walker_i*2*N) + 2*n]   = TR[(0*3 + 1)];
	data21[(walker_i*2*N) + 2*n]   = TR[(1*3 + 0)];
	data31[(walker_i*2*N) + 2*n]   = TR[(2*3 + 0)];
	data12[(walker_i*2*N) + 2*n+1] = TI[(0*3 + 1)];
	data21[(walker_i*2*N) + 2*n+1] = TI[(1*3 + 0)];
	data31[(walker_i*2*N) + 2*n+1] = TI[(2*3 + 0)];
	data13[(walker_i*2*N) + 2*n]   = TR[(0*3 + 2)];
	data23[(walker_i*2*N) + 2*n]   = TR[(1*3 + 2)];
	data32[(walker_i*2*N) + 2*n]   = TR[(2*3 + 1)];
	data13[(walker_i*2*N) + 2*n+1] = TI[(0*3 + 2)];
	data23[(walker_i*2*N) + 2*n+1] = TI[(1*3 + 2)];
	data32[(walker_i*2*N) + 2*n+1] = TI[(2*3 + 1)];

	return;
}


__global__
void GenWave(Waveform *wfm_trans, int N, int nwalkers,
						 double *data12, double *data21, double *data13,
						 double *data31, double *data23, double *data32){
	double t=0.0;
	Waveform *wfm;
	int tid = (int)threadIdx.x;

	__shared__ double kdotr[9*256];
	double TR[9];
	double TI[9];
	double dplus[9];
	double dcross[9];
	double x[3], y[3], z[3];
	double xi[3], fonfs[3];
	double r12[3], r21[3], r13[3], r31[3], r23[3], r32[3];

	for (int walker_i = blockIdx.y * blockDim.y + threadIdx.y;
			 walker_i < nwalkers;
			 walker_i += blockDim.y * gridDim.y){

	wfm = &wfm_trans[walker_i];

	for (int n = blockIdx.x * blockDim.x + threadIdx.x;
			 n < N;
			 n += blockDim.x * gridDim.x){

				 t = wfm->T*(double)(n)/(double)N;
				 calc_xi_f(wfm ,t, n, N, x, y, z, xi, fonfs);		  // calc frequency and time variables
				 calc_sep_vecs(wfm, n, N, x, y, z, r12, r21, r13, r31, r23, r32);       // calculate the S/C separation vectors
				 calc_d_matrices(wfm, n, N, dplus, dcross, r12, r21, r13, r31, r23, r32);     // calculate pieces of waveform
				 calc_kdotr(wfm, n, N, &kdotr[tid*9], r12, r21, r13, r31, r23, r32);		  // calculate dot product
				 get_transfer(wfm, t, n, N, &kdotr[tid*9], TR, TI, dplus, dcross, xi, fonfs);     // Calculating Transfer function
				 fill_time_series(walker_i, n, N, TR, TI, data12, data21, data13, data31, data23, data32); // Fill  time series data arrays with slowly evolving signal.
		}
}

}

// cuFFT API errors
static const char *_cudaGetErrorEnum(hipfftResult error)
{
    switch (error)
    {
        case HIPFFT_SUCCESS:
            return "HIPFFT_SUCCESS";

        case HIPFFT_INVALID_PLAN:
            return "HIPFFT_INVALID_PLAN";

        case HIPFFT_ALLOC_FAILED:
            return "HIPFFT_ALLOC_FAILED";

        case HIPFFT_INVALID_TYPE:
            return "HIPFFT_INVALID_TYPE";

        case HIPFFT_INVALID_VALUE:
            return "HIPFFT_INVALID_VALUE";

        case HIPFFT_INTERNAL_ERROR:
            return "HIPFFT_INTERNAL_ERROR";

        case HIPFFT_EXEC_FAILED:
            return "HIPFFT_EXEC_FAILED";

        case HIPFFT_SETUP_FAILED:
            return "HIPFFT_SETUP_FAILED";

        case HIPFFT_INVALID_SIZE:
            return "HIPFFT_INVALID_SIZE";

        case HIPFFT_UNALIGNED_DATA:
            return "HIPFFT_UNALIGNED_DATA";
    }

    return "<unknown>";
}

#define cufftSafeCall(err)      __cufftSafeCall(err, __FILE__, __LINE__)
inline void __cufftSafeCall(hipfftResult err, const char *file, const int line)
{
    if( HIPFFT_SUCCESS != err) {
    fprintf(stderr, "CUFFT error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n",__FILE__, __LINE__,err,
                                _cudaGetErrorEnum(err));
    hipDeviceReset(); assert(0);
	}
}


void fft_data(double *data12, double *data21, double *data13, double *data31, double *data23, double *data32, hipfftHandle plan, int nwalkers)
{

	cufftSafeCall(hipfftExecZ2Z(plan, (hipfftDoubleComplex*)data12, (hipfftDoubleComplex*)data12, -1));
	hipDeviceSynchronize();
	gpuErrchk(hipGetLastError());

	if (hipfftExecZ2Z(plan, (hipfftDoubleComplex*)data21, (hipfftDoubleComplex*)data21, -1) != HIPFFT_SUCCESS){
	fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
	return;}
	hipDeviceSynchronize();
	gpuErrchk(hipGetLastError());

	if (hipfftExecZ2Z(plan, (hipfftDoubleComplex*)data31, (hipfftDoubleComplex*)data31, -1) != HIPFFT_SUCCESS){
	fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
	return;}
	hipDeviceSynchronize();
	gpuErrchk(hipGetLastError());

	if (hipfftExecZ2Z(plan, (hipfftDoubleComplex*)data13, (hipfftDoubleComplex*)data13, -1) != HIPFFT_SUCCESS){
	fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
	return;}
	hipDeviceSynchronize();
	gpuErrchk(hipGetLastError());

	if (hipfftExecZ2Z(plan, (hipfftDoubleComplex*)data23, (hipfftDoubleComplex*)data23, -1) != HIPFFT_SUCCESS){
	fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
	return;}
	hipDeviceSynchronize();
	gpuErrchk(hipGetLastError());

	if (hipfftExecZ2Z(plan, (hipfftDoubleComplex*)data32, (hipfftDoubleComplex*)data32, -1) != HIPFFT_SUCCESS){
	fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
	return;}
	hipDeviceSynchronize();
	gpuErrchk(hipGetLastError());

	return;
}

__global__
void unpack_data_1(Waveform *wfm_trans, double *data12, double *data21, double *data13,
double *data31, double *data23, double *data32, int nwalkers)
{

	Waveform *wfm;
	for (int walker_i = blockIdx.y * blockDim.y + threadIdx.y;
			 walker_i < nwalkers;
			 walker_i += blockDim.y * gridDim.y){

		wfm = &wfm_trans[walker_i];
		int N = wfm->N;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
			 i < N;
			 i += blockDim.x * gridDim.x){
		// populate from most negative (Nyquist) to most positive (Nyquist-1)
		wfm->a12[i]   = 0.5*data12[(walker_i*2*N) + N+i]/(double)N;  // moved the 0.5
		wfm->a21[i]   = 0.5*data21[(walker_i*2*N) + N+i]/(double)N;
		wfm->a31[i]   = 0.5*data31[(walker_i*2*N) + N+i]/(double)N;
		wfm->a12[i+N] = 0.5*data12[(walker_i*2*N) + i]/(double)N;
		wfm->a21[i+N] = 0.5*data21[(walker_i*2*N) + i]/(double)N;
		wfm->a31[i+N] = 0.5*data31[(walker_i*2*N) + i]/(double)N;
		wfm->a13[i]   = 0.5*data13[(walker_i*2*N) + N+i]/(double)N;
		wfm->a23[i]   = 0.5*data23[(walker_i*2*N) + N+i]/(double)N;
		wfm->a32[i]   = 0.5*data32[(walker_i*2*N) + N+i]/(double)N;
		wfm->a13[i+N] = 0.5*data13[(walker_i*2*N) + i]/(double)N;
		wfm->a23[i+N] = 0.5*data23[(walker_i*2*N) + i]/(double)N;
		wfm->a32[i+N] = 0.5*data32[(walker_i*2*N) + i]/(double)N;
	}
}
}

/*
__global__
void unpack_data_2(Waveform *wfm_trans, int nwalkers)
{
	//   Renormalize so that the resulting time series is real

	Waveform *wfm;
	for (int walker_i = blockIdx.y * blockDim.y + threadIdx.y;
			 walker_i < nwalkers;
			 walker_i += blockDim.y * gridDim.y){

		wfm = &wfm_trans[walker_i];
		int N = wfm->N;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
			 i < 2*N;
			 i += blockDim.x * gridDim.x)
	{
		wfm->d[0*3*2*N + 1*2*N + i] = 0.5*wfm->a12[i];
		wfm->d[1 *3*2*N + 0 *2*N + i] = 0.5*wfm->a21[i];
		wfm->d[2 *3*2*N + 0 *2*N + i] = 0.5*wfm->a31[i];
		wfm->d[0*3*2*N + 2*2*N + i] = 0.5*wfm->a13[i];
		wfm->d[1 *3*2*N + 2 *2*N + i] = 0.5*wfm->a23[i];
		wfm->d[2 *3*2*N + 1 *2*N + i] = 0.5*wfm->a32[i];
	}
}
}
*/


__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long* address_as_ull =
                              (unsigned long long*)address;
    unsigned long long old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}


__device__
void XYZ(int i, double *a12, double *a21, double *a13, double *a31, double *a23, double *a32, double f0, long q, long M, double dt, double Tobs, double *XLS_r, double *YLS_r, double *ZLS_r,
					double* XSL_r, double* YSL_r, double* ZSL_r, double *XLS_i, double *YLS_i, double *ZLS_i, double *XSL_i, double *YSL_i, double *ZSL_i)
{
	double fonfs;
	double c3, s3, c2, s2, c1, s1;
	double f;
	double phiLS, cLS, sLS, phiSL, cSL, sSL;

	double X_1, X_2, Y_1, Y_2, Z_1, Z_2;

	// YLS = malloc(2*M*sizeof(double));
	// ZLS = malloc(2*M*sizeof(double));

	phiLS = PI2*f0*(dt/2.0-Larm/C);

	cLS = cos(phiLS);
	sLS = sin(phiLS);

	//double phiLS = 2.0*pi*f0*(dt/2.0-L/clight);
	//double cLS = cos(phiLS); double sLS = sin(phiLS);

	phiSL = M_PI/2.0-2.0*M_PI*f0*(Larm/C);
	cSL = cos(phiSL);
	sSL = sin(phiSL);

  //printf("Stas, q=%ld, f0=%f, check: %f, %f \n", q, f0, q/Tobs, Tobs);

		f = ((double)(q + i - M/2))/Tobs;
		//if (i == 0){
		//		double f1 = ((double)(q + i -1 - M/2))/Tobs;
		//		double f2 = ((double)(q + i - M/2))/Tobs;
				//printf("%e, %e, %ld, %ld, %ld\n", f, f2 - f1, q, i, M/2);
		//}
		fonfs = f/fstar;
		//printf("Stas fonfs = %f, %f, %f, %f \n", fonfs, f, fstar, Tobs);
		c3 = cos(3.*fonfs);  c2 = cos(2.*fonfs);  c1 = cos(1.*fonfs);
		s3 = sin(3.*fonfs);  s2 = sin(2.*fonfs);  s1 = sin(1.*fonfs);

		X_1   = (a12[2*i]-a13[2*i])*c3 + (a12[2*i+1]-a13[2*i+1])*s3 +
		           (a21[2*i]-a31[2*i])*c2 + (a21[2*i+1]-a31[2*i+1])*s2 +
		           (a13[2*i]-a12[2*i])*c1 + (a13[2*i+1]-a12[2*i+1])*s1 +
		           (a31[2*i]-a21[2*i]);

		X_2 = (a12[2*i+1]-a13[2*i+1])*c3 - (a12[2*i]-a13[2*i])*s3 +
		           (a21[2*i+1]-a31[2*i+1])*c2 - (a21[2*i]-a31[2*i])*s2 +
		           (a13[2*i+1]-a12[2*i+1])*c1 - (a13[2*i]-a12[2*i])*s1 +
		           (a31[2*i+1]-a21[2*i+1]);

		Y_1   = (a23[2*i]-a21[2*i])*c3 + (a23[2*i+1]-a21[2*i+1])*s3 +
		           (a32[2*i]-a12[2*i])*c2 + (a32[2*i+1]-a12[2*i+1])*s2+
		           (a21[2*i]-a23[2*i])*c1 + (a21[2*i+1]-a23[2*i+1])*s1+
		           (a12[2*i]-a32[2*i]);

		Y_2 = (a23[2*i+1]-a21[2*i+1])*c3 - (a23[2*i]-a21[2*i])*s3+
		           (a32[2*i+1]-a12[2*i+1])*c2 - (a32[2*i]-a12[2*i])*s2+
		           (a21[2*i+1]-a23[2*i+1])*c1 - (a21[2*i]-a23[2*i])*s1+
		           (a12[2*i+1]-a32[2*i+1]);

		Z_1   = (a31[2*i]-a32[2*i])*c3 + (a31[2*i+1]-a32[2*i+1])*s3+
		           (a13[2*i]-a23[2*i])*c2 + (a13[2*i+1]-a23[2*i+1])*s2+
		           (a32[2*i]-a31[2*i])*c1 + (a32[2*i+1]-a31[2*i+1])*s1+
		           (a23[2*i]-a13[2*i]);

		Z_2 = (a31[2*i+1]-a32[2*i+1])*c3 - (a31[2*i]-a32[2*i])*s3+
		           (a13[2*i+1]-a23[2*i+1])*c2 - (a13[2*i]-a23[2*i])*s2+
		           (a32[2*i+1]-a31[2*i+1])*c1 - (a32[2*i]-a31[2*i])*s1+
		           (a23[2*i+1]-a13[2*i+1]);

		// XLS_r   =  (X_1*cLS - X_2*sLS);
		// XLS_i = -(X_1*sLS + X_2*cLS);
		// YLS_r   =  (Y_1*cLS - Y_2*sLS);
		// YLS_i = -(Y_1*sLS + Y_2*cLS);
		// ZLS_r   =  (Z_1*cLS - Z_2*sLS);
		// ZLS_i = -(Z_1*sLS + Z_2*cLS);
    //
		// XSL_r   =  2.0*fonfs*(X_1*cSL - X_2*sSL);
		// XSL_i = -2.0*fonfs*(X_1*sSL + X_2*cSL);
		// YSL_r   =  2.0*fonfs*(Y_1*cSL - Y_2*sSL);
		// YSL_i = -2.0*fonfs*(Y_1*sSL + Y_2*cSL);
		// ZSL_r   =  2.0*fonfs*(Z_1*cSL - Z_2*sSL);
		// ZSL_i = -2.0*fonfs*(Z_1*sSL + Z_2*cSL);

		// Alternative polarization definition
		*XLS_r   =  (X_1*cLS - X_2*sLS);
		*XLS_i =  (X_1*sLS + X_2*cLS);
		*YLS_r   =  (Y_1*cLS - Y_2*sLS);
		*YLS_i =  (Y_1*sLS + Y_2*cLS);
		*ZLS_r   =  (Z_1*cLS - Z_2*sLS);
		*ZLS_i =  (Z_1*sLS + Z_2*cLS);

		*XSL_r   =  2.0*fonfs*(X_1*cSL - X_2*sSL);
		*XSL_i =  2.0*fonfs*(X_1*sSL + X_2*cSL);
		*YSL_r   =  2.0*fonfs*(Y_1*cSL - Y_2*sSL);
		*YSL_i =  2.0*fonfs*(Y_1*sSL + Y_2*cSL);
		*ZSL_r   =  2.0*fonfs*(Z_1*cSL - Z_2*sSL);
		*ZSL_i =  2.0*fonfs*(Z_1*sSL + Z_2*cSL);

	// for(i=0; i<2*M; i++)
	// {
	// 	// A channel
	// 	ALS[i] = (2.0*XLS[i] - YLS[i] - ZLS[i])/3.0;
	// 	// E channel
	// 	ELS[i] = (ZLS[i]-YLS[i])/SQ3;
	// }


	//free(YLS);
	//free(ZLS);

	return;
}

__global__
void XYZ_wrap(Waveform *wfm_trans, int nwalkers, long M, double dt, double Tobs, double *XLS, double *YLS, double *ZLS){

		int N;
			long add_ind;
		double asd1, asd2, asd3;

		Waveform *wfm;
		for (int walker_i = blockIdx.y * blockDim.y + threadIdx.y;
				 walker_i < nwalkers;
				 walker_i += blockDim.y * gridDim.y){

		wfm = &wfm_trans[walker_i];
		N = wfm->N;
		for (int i = blockIdx.x * blockDim.x + threadIdx.x;
				 i < M;
				 i += blockDim.x * gridDim.x)
		{


		double XLS_r, YLS_r, ZLS_r, XSL_r, YSL_r, ZSL_r, XLS_i, YLS_i, ZLS_i, XSL_i, YSL_i, ZSL_i;

		XYZ(i, wfm->a12, wfm->a21, wfm->a13, wfm->a31, wfm->a23, wfm->a32, wfm->params[0]/wfm->T, wfm->q, N, dt, Tobs,
				&XLS_r, &YLS_r, &ZLS_r, &XSL_r, &YSL_r, &ZSL_r, &XLS_i, &YLS_i, &ZLS_i, &XSL_i, &YSL_i, &ZSL_i);

		add_ind = (wfm->q + i - M/2);

		atomicAddDouble(&XLS[2*add_ind], XLS_r/asd1);
		atomicAddDouble(&XLS[2*add_ind+1], XLS_i/asd1);

		atomicAddDouble(&YLS[2*add_ind], YLS_r/asd2);
		atomicAddDouble(&YLS[2*add_ind+1], YLS_i/asd2);

		atomicAddDouble(&ZLS[2*add_ind], ZLS_r/asd3);
		atomicAddDouble(&ZLS[2*add_ind+1], ZLS_i/asd3);

		/*atomicAddDouble(&XSL[2*add_ind], XSL_r/asd1);
		atomicAddDouble(&XSL[2*add_ind+1], XSL_i)/asd1;

		atomicAddDouble(&YSL[2*add_ind], YSL_r/asd2);
		atomicAddDouble(&YSL[2*add_ind+1], YSL_i/asd2);

		atomicAddDouble(&YSL[2*add_ind], ZSL_r/asd3);
		atomicAddDouble(&ZSL[2*add_ind+1], ZSL_i/asd3);*/


}
}
}

/*
__global__
void prep_with_ASD(double *XLS, double *YLS, double *ZLS, double *channel1_ASDinv, double *channel2_ASDinv, double *channel3_ASDinv, int data_stream_length){
	double asd1, asd2, asd3;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
			 i < data_stream_length;
			 i += blockDim.x * gridDim.x)
	{
			asd1 = channel1_ASDinv[i];
			asd2 = channel2_ASDinv[i];
			asd3 = channel3_ASDinv[i];

			XLS[2*i] = XLS[2*i]/asd1;
			XLS[2*i+1] = XLS[2*i+1]/asd1;

			YLS[2*i] = YLS[2*i]/asd2;
			YLS[2*i+1] = YLS[2*i+1]/asd2;

			ZLS[2*i] = ZLS[2*i]/asd3;
			ZLS[2*i+1] = ZLS[2*i+1]/asd3;
	}
}
*/
